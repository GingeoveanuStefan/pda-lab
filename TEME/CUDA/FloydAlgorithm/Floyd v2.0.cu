#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

__global__ void Floyd(float* A, int N, int k) {
    int i = blockIdx.x;                             //block
    int j = blockDim.x * blockIdx.x + threadIdx.x;  //thread

    if( (i<N && j<N) && (a[i][k]+a[k][j]) < a[i][j] ) {
        a[i][j] = a[i][k]+a[k][j];
    }
}

int main() {
    FILE *file;
    file = fopen("floyd.txt","r");

    int N;
    float *h_A, *d_A;
    int i, j, k;
    size_t sizeA;

    fscanf(file, "%d%", &N);
    sizeA = N * N * sizeof(float);

    h_A = (float*)malloc(sizeA);

    // INPUT MATRIX
    for(i=0 ;i<M; i++) {
        for(j=0; j<N; j++) {
            fscanf(file, "%f%", &A[i][j]);
        }
    }

    fclose(file);

    // Copy data to device
    hipMalloc(&d_A, sizeA);
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);

    // Invoke kernel

    int numOfThreads = 256;
	int numOfBlocks = (N + numOfThreads - 1) / numOfThreads;
	
	dim3 dimBlock(numOfThreads, numOfThreads);
	dim3 dimGrid(numOfBlocks, numOfBlocks);
	
    for(k=0; k<N; k++) {
		Floyd <<<dimGrid, dimBlock>>>(d_A, N, k);
	}

    // COPY RESULT TO HOST
    hipMemcpy(h_A, d_A, sizeA, hipMemcpyDeviceToHost);

    hipFree(d_A);

    //PRINT RESULT
    for(i=0; i<N; i++){
        for(j=0; j<N;j++){
            printf("%f ", d_A[i][j]);
        }
        printf("/n");
    }

    free(h_A);

    return 0;
}
