#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

__global__ void Floyd(float* A, int N) {
    int i = blockIdx.x;                             //block
    int j = blockDim.x * blockIdx.x + threadIdx.x;  //thread
    int k;

    for(k = 0; k < N; k++) {
        if( (a[i][k]+a[k][j]) < a[i][j] )
            a[i][j] = a[i][k]+a[k][j];
    }
}

int main() {
    FILE *file;
    file = fopen("floyd.txt","r");

    int N;
    float *h_A;
    float *d_A;
    int i, j;
    size_t sizeA;

    fscanf(file, "%d%", &N);
    sizeA = N * N * sizeof(float);

    h_A = (float*)malloc(sizeA);

    // INPUT MATRIX
    for(i=0 ;i<M; i++) {
        for(j=0; j<N; j++) {
            fscanf(file, "%f%", &A[i][j]);
        }
    }

    fclose(file);

    hipMalloc(&d_A, sizeA);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);

    // Invoke kernel with N blocks and N threads per block
    Floyd<<<N, N>>>(d_A, N);

    // COPY RESULT TO HOST
    hipMemcpy(h_A, d_A, sizeA, hipMemcpyDeviceToHost);

    hipFree(d_A);

    //PRINT RESULT
    for(i=0; i<N; i++){
        for(j=0; j<N;j++){
            printf("%f ", d_A[i][j]);
        }
        printf("/n");
    }

    free(h_A);

    return 0;
}
