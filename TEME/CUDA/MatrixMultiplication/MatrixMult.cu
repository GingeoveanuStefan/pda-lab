#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

__global__ void MatrixMult(float* A, float* B, float* C, int N) {
    int i = blockIdx.x;                             //block
    int j = blockDim.x * blockIdx.x + threadIdx.x;  //thread
    int k;

    for(k = 0; k < N; k++)
        C[i][j] += A[i][k] * B[k][j];
}

int main() {
    FILE *file;
    file = fopen("input.txt","r");

    int M, N, P;
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    int i, j;
    size_t sizeA, sizeB, sizeC;

    fscanf(file, "%d%", &M);
    fscanf(file, "%d%", &N);
    fscanf(file, "%d%", &P);

    sizeA = M * N * sizeof(float);
    sizeB = N * P * sizeof(float);
    sizeC = M * P * sizeof(float);

    h_A = (float*)malloc(sizeA);
    h_B = (float*)malloc(sizeB);
    h_C = (float*)malloc(sizeC);

    // Read matrices
    for(i=0 ;i<M; i++) {    // A
        for(j=0; j<N; j++) {
            fscanf(file, "%f%", &A[i][j]);
        }
    }
    for(i=0 ;i<N; i++) {    // B
        for(j=0; j<P; j++) {
            fscanf(file, "%f%", &A[i][j]);
        }
    }

    fclose(file);

    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    // Invoke kernel with M blocks (one block per line) and P threads per block (one thread per element)
    MatrixMult<<<M, P>>>(d_A, d_B, d_C, N);

    // COPY RESULT TO HOST
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    //PRINT RESULT
    for(i=0; i<M; i++){
        for(j=0; j<P;j++){
            printf("%f ", d_C[i][j]);
        }
        printf("/n");
    }

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
