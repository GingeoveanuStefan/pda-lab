#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

__global__ void MatrixMult(int*** img, int*** med) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(row == 0 || col == 0 || row == (blockDim.y-1) || col == (blockDim.x-1) ) {
		med[row][col][0] = img[row][col][0];	//R
		med[row][col][1] = img[row][col][1];	//G
		med[row][col][2] = img[row][col][2];	//B
	}
	else {
		med[row][col][0] = 	( 	//R
			img[row-1][col-1][0] + img[row-1][col][0] + img[row-1][col+1][0] +
			img[row][col-1][0]   + img[row][col][0]   + img[row][col+1][0]   +
			img[row+1][col-1][0] + img[row+1][col][0] + img[row+1][col+1][0] ) / 9;
			
		med[row][col][1] = 	( 	//G
			img[row-1][col-1][1] + img[row-1][col][1] + img[row-1][col+1][1] +
			img[row][col-1][1]   + img[row][col][1]   + img[row][col+1][1]   +
			img[row+1][col-1][1] + img[row+1][col][1] + img[row+1][col+1][1] ) / 9;
			
		med[row][col][2] = 	( 	//B
			img[row-1][col-1][2] + img[row-1][col][2] + img[row-1][col+1][2] +
			img[row][col-1][2]   + img[row][col][2]   + img[row][col+1][2]   +
			img[row+1][col-1][2] + img[row+1][col][2] + img[row+1][col+1][2] ) / 9;
	}
}

int main() {
    FILE *file;
    file = fopen("image.txt","r");

	//img[height][width][pixel]
	
    int height, width;
    int ***img;
    int ***org, ***med;
    int i,j,k;
    size_t size;

	//READ THE IMAGE
    fscanf(file, "%d%", &height);
	fscanf(file, "%d%", &width);
	
    size = height * width * 3 * sizeof(int);
    img = (float*)malloc(size);
    for(i=0 ;i<height; i++) {
        for(j=0; j<width; j++) {
			for(k=0; k<3; k++) {	//RGB
				fscanf(file, "%f%", &img[i][j][k]);
			}
        }
    }
    fclose(file);
	

	
	//Alloc cuda matrices
    hipMalloc(&org, size);	//original file
    hipMalloc(&med, size);	//final output file

    hipMemcpy(org, img, size, hipMemcpyHostToDevice);
    hipMemcpy(med, img, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int numBlocks = 1;
	dim3 numThreads(height, width);
	
	MedianFilter<<<numBlocks, numThreads>>>(org,med);

	
    // COPY RESULT TO HOST
    hipMemcpy(img, med, size, hipMemcpyDeviceToHost);
    hipFree(org);
    hipFree(med);

    //PRINT RESULT
    for(i=0; i<height; i++){
        for(j=0; j<width;j++){
			for(k=0; k<width;k++){
				printf("%d,", img[i][j][k]);
			}
			printf("   ");
        }
        printf("/n");
    }
	
    free(img);

    return 0;
}