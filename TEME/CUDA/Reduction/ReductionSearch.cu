#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

// g_data		=	array of values
// target		=	value to find
// g_posdata	=	array of array of positions
//		g_posdata[0]	has N/1 + 1 max elements
//		g_posdata[N/2]	has N/2 + 1 max elements
//		g_posdata[N/4]	has N/4  max elements
//		g_posdata[N/8]	has N/8  max elements
//		g_posdata[N/16]	has N/16 max elements


__global__ void reduce_sum(int *g_data, int **g_posdata, int target) {
	extern __shared__ int sdata[][];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x *blockDim.x + threadIdx.x;
	
	if(g_data[i] == target) {	//add initial position if target found
		sdata[tid][0] = i;
	}
	
	__syncthreads();
	
	// do reduction in shared mem
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0)  {
			if (g_data[tid + s] == target ){	//found target on paired position
				unsigned int p;
				for(p = 0; p<blockDim.x && sdata[tid][p] != -1 ; p++);	//go to end positions list
				sdata[tid][p] = tid + s;				// and add position
			}
		}
		__syncthreads();
	}
	
	// write result for this block to global mem
	if (tid == 0) {
		g_posdata[blockIdx.x] = sdata[0];
	}
}

int main(){
	FILE *file;
    file = fopen("input.txt","r");
	
	
	int *h_data, *d_data;
	int **h_posdata, **d_posdata;
	int n, target;
	
	size_t size, pos_size;
	
	//Read data
    fscanf(file, "%d%", &n);
	
	//Input values
    size = n * sizeof(int);
    h_data = (float*)malloc(size);
    for(i=0 ;i<n; i++) {
		fscanf(file, "%d%", &h_data[i]);
    }
	
	fscanf(file, "%d%", &target);
	
    fclose(file);
	
	pos_size = n * n * sizeof(int);
	h_posdata = (float*)malloc(pos_size);
	
	// copy inputs to device
	hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
	hipMemcpy(d_posdata, h_posdata, pos_size, hipMemcpyHostToDevice);

    int numBlocks = 1;
	dim3 numThreads(n);

	reduce_sum <<< numBlocks, numThreads >>> (d_data, d_posdata, target);

	hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_posdata, d_posdata, pos_size, hipMemcpyDeviceToHost);

    for(i=0; i<n && h_posdata[0][i] != -1; i++) {
		fscanf(file, "%d%", &h_posdata[0][i]);
    }

	// free device memory
	hipFree(d_data);
	hipFree(d_posdata);

	// free host memory
	free(h_data);
	free(h_posdata);

	return 0;
}